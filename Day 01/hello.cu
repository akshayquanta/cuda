
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void helloCUDA() {
    printf("Hello from CUDA kernel!\n");
}

int main() {
    helloCUDA<<<1, 1>>>(); // Launch kernel with 1 block and 1 thread
    hipDeviceSynchronize(); // Ensure kernel execution completes
    return 0;
}
